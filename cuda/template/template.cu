#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

using namespace std;

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(float *g_idata, float *g_odata)
{
    // shared memory
    // the size is determined by the host application
    extern  __shared__  float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x;

    // read in input data from global memory
    sdata[tid] = g_idata[tid];
    __syncthreads();

    // perform some computations
    sdata[tid] = (float) num_threads * sdata[tid];
    __syncthreads();

    // write data to global memory
    g_odata[tid] = sdata[tid];
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate and initalize host memory
    float *h_idata = (float *) malloc(mem_size);
    for (unsigned int i = 0; i < num_threads; ++i) h_idata[i] = (float) i;
    for (size_t i = 0; i < num_threads; i++) cout << h_idata[i] << " "; cout << endl;

    // allocate and copy device memory
    float *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size,
                               hipMemcpyHostToDevice));

    // allocate device memory for result
    float *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // setup and execute
    dim3  grid(1, 1, 1);
    dim3  threads(num_threads, 1, 1);
    testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);
    getLastCudaError("Kernel execution failed");

    // allocate and copy the result from device to host
    float *h_odata = (float *) malloc(mem_size);
    checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads,
                               hipMemcpyDeviceToHost));
    for (size_t i = 0; i < num_threads; i++) cout << h_odata[i] << " "; cout << endl;

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    // compute reference solution and check
    float *reference = (float *) malloc(mem_size);
    computeGold(reference, h_idata, num_threads); // Matrix-Matrix product
    bTestResult = compareData(reference, h_odata, num_threads, 0.0f, 0.0f);

    // cleanup memory
    free(h_idata);
    free(h_odata);
    free(reference);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits   
    hipDeviceReset();
    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
